#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "set_device.cuh"

__device__ int get_global_index(void)
{
  return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ int get_constant(void)
{
  return 7;
}

__global__ void kernel1(int *array)
{
  int index = get_global_index();
  array[index] = get_constant();
}

__global__ void kernel2(int *array)
{
  int index = get_global_index();
  array[index] = get_global_index();
}

int main(int argc, char* argv[])
{
  if (argc != 2) {
    printf("Usage:\n./device_functions_main [device_number]\n");
    return 1;
  }
  set_device(argv);
 
  int num_elements = 256;
  int num_bytes = num_elements * sizeof(int);

  int *device_array = 0;
  int *host_array = 0;

  // allocate memory
  host_array = (int*)malloc(num_bytes);
  hipMalloc((void**)&device_array, num_bytes);

  int block_size = 128;
  int grid_size = num_elements / block_size;

  // launch kernel1 and inspect its results
  kernel1<<<grid_size,block_size>>>(device_array);
  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  printf("kernel1 results:\n");
  for(int i = 0; i < num_elements; ++i)
  {
    printf("%d ", host_array[i]);
  }
  printf("\n\n");

  // launch kernel2 and inspect its results
  kernel2<<<grid_size,block_size>>>(device_array);
  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  printf("kernel2 results:\n");
  for(int i = 0; i < num_elements; ++i)
  {
    printf("%d ", host_array[i]);
  }
  printf("\n\n");

  // deallocate memory
  free(host_array);
  hipFree(device_array);
  return 0;
}