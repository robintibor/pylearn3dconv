#include "hip/hip_runtime.h"
#include <stdio.h>
#include "set_device.cuh"

__global__ void foo()
{
}

int main(int argc, char* argv[])
{
  if (argc != 2) {
    printf("Usage:\n./tutorial_start_main [device_number]\n");
    return 1;
  }
  set_device(argv);
  int num_elements = 16;
  int num_bytes = num_elements * sizeof(int);

  printf("Size of int: %zu\n", sizeof(int));
  int *device_array = 0;
  int *host_array = 0;

  // malloc host memory
  host_array = (int*)malloc(num_bytes);

  // hipMalloc device memory
  hipMalloc((void**)&device_array, num_bytes);

  // zero out the device array with hipMemset
  hipMemset(device_array, 0, num_bytes);

  // copy the contents of the device array to the host
  hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

  // print out the result element by element
  for(int i = 0; i < num_elements; ++i)
    printf("%d ", host_array[i]);
  printf("\n");
  // use free to deallocate the host array
  free(host_array);

  // use hipFree to deallocate the device array
  hipFree(device_array);
  
  return 0;
}